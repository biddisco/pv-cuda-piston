#include "hip/hip_runtime.h"
#ifdef _WIN32
 #include <windows.h>
#endif

#include <iostream>

// include before gl_interop
#include "vtkgl.h"
//
// CUDA
//
#include <cuda_gl_interop.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>

//
// Thrust
//
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>

//
// Piston
//
//#include "piston/piston_math.h"
//
#include "vtkPistonDataObject.h"
#include "vtkPistonDataWrangling.h"
#include "vtkPistonReference.h"

#include "../vtkTwoScalarsToColorsPainter.h"

#define USE_FLOAT_FOR_DEPTH_SORT

namespace vtkpiston {

inline __host__ __device__ float dot(float3 a, float3 b)
{ 
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

inline __host__ __device__ float dot(double3 a, double3 b)
{ 
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
typedef thrust::device_vector<float>::iterator FloatIterator;
typedef thrust::tuple<FloatIterator, FloatIterator> FloatIteratorTuple;
typedef thrust::tuple<float&, float&> FloatTuple;
typedef thrust::zip_iterator<FloatIteratorTuple> Float2Iterator;
//typedef thrust::detail::tuple_of_iterator_references<float &, float &, thrust::null_type, thrust::null_type, thrust::null_type, thrust::null_type, thrust::null_type, thrust::null_type, thrust::null_type, thrust::null_type> TupleDef;

static __inline__ __host__ __device__ double3 make_double3(const float3 &f3)
{
  double3 t; t.x = f3.x; t.y = f3.y; t.z = f3.z; return t;
}

static __inline__ __host__ __device__ double3 make_double3(const float  &x, const float  &y, const float  &z)
{
  double3 t; t.x = x; t.y = y; t.z = z; return t;
}

//------------------------------------------------------------------------------
// The colour map struct is templated (T) over the iterator type that we will use
// in our transform function.
//
// Because we use complicated zip/tuple iterators which are partially dereferenced
// by the tuple/iterator code inside the transform call, we separately template
// the operator() over the final dereferenced (element) type that will be passed
// to the operator().
//
// Two specializations are provided for color tables with/without vertex opacity
// 1) scalars        : T=float(iterator), element=float 
// 2) scalar/opacity : T=FloatIteratorTuple(iterator), element=FloatTuple 

struct color_map 
{
  const float min;
  const float max;
  const int   size;
  unsigned char *table;
  float       alpha;
  float      *opacity;
  float      *scalars;

  color_map(unsigned char *RGBtable, size_t arrSize, float rMin, float rMax, double a, float *scalararray, float *opacityarray) :
    min(rMin),
    max(rMax),
    size((arrSize / 3) - 1),
    table(RGBtable),
    alpha(a),
    scalars(scalararray),
    opacity(opacityarray)
    {
    }
//  const float inv = 0.0039215686274509803921568627451;
  // the internal calculation which is independent of template types
  __host__ __device__ inline uchar4 calc(float val, float opac) 
  { 
    // convert val to lookuptable index
    int index = 0;
    if ((max - min) > 0.0) {
      index = ( (val - min) / (max - min) ) * size;
    }
    if (index < 0)    index = 0;
    if (index > size) index = size;
    // convert to RGB tuple index
    // 1/255 = 0.0039215686274509803921568627451
    index *= 3; 
    return make_uchar4(table[index]*opac, table[index + 1]*opac, table[index + 2]*opac, 255.0*opac);
  };

  __host__ __device__ uchar4 color_map::operator()(float t)
  {
    float val = t;
    return calc(val, alpha);
  }

//  __host__ __device__ float4 color_map::operator()(const TupleDef &t)
//  {
//    float val  = thrust::get<0>(t);
//    float opac = thrust::get<1>(t)*alpha;
//    return calc(val, opac);
//  }

};

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
int GetCudaDeviceCount()
{
  int num;
  hipGetDeviceCount(&num);
  return num;
}
//------------------------------------------------------------------------------
void CudaGLInit(int device)
{
//  hipDeviceProp_t prop;
    // Fill it with zeros
//  memset(&prop,0,sizeof(hipDeviceProp_t));
    // Pick a GPU capable of 1.0 or better
//  prop.major=1; prop.minor=0;
//  hipChooseDevice(&dev,&prop);
 
  hipSetDevice(device);

  // Set OpenGL device
  hipError_t res = cudaGLSetGLDevice(device);

  if (res != hipSuccess)
    {
    std::cerr << "Set device failed  ... " << hipGetErrorString(res) << endl;
    return;
    }
}

//------------------------------------------------------------------------------
void CudaRegisterBuffer(struct hipGraphicsResource **vboResource,
                        GLuint vboBuffer)
{
  hipError_t res =
    hipGraphicsGLRegisterBuffer(vboResource, vboBuffer,
                                cudaGraphicsMapFlagsWriteDiscard);
  if (res != hipSuccess)
  {
    std::cerr << "Register buffer failed ... " << hipGetErrorString(res) << endl;
    return;
  }
}

//------------------------------------------------------------------------------
// Compute the distance from the camera to a single point
//------------------------------------------------------------------------------
template <typename T>
struct distance_functor 
{
  T cameravector;

  // construct with a constant camera vector
  __host__ __device__ distance_functor(T &cam) : cameravector(cam) {}

  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<1>(t) = dot(thrust::get<0>(t), cameravector);  
  }
};
//------------------------------------------------------------------------------
// Takes 3 point distances and computes the mean (i.e. triangular cell) distance
//------------------------------------------------------------------------------
template <typename T>
struct celldistance_functor 
{
  const T *vertex_distances;
  
  // construct with a precomputed distance vector for every vertex
  __host__ __device__ celldistance_functor(T *v) : vertex_distances(v) {}
  
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  { 
    thrust::get<1>(t) = (vertex_distances[thrust::get<0>(t).x] + 
                         vertex_distances[thrust::get<0>(t).y] +
                         vertex_distances[thrust::get<0>(t).z])/3.0;
  }
};

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
void DepthSortPolygons(vtkPistonDataObject *id, double *cameravec, int direction)
{
  vtkPistonReference *tr = id->GetReference();
  if (tr->type != VTK_POLY_DATA || tr->data == NULL) {
    // Type mismatch, don't bother trying
    return;
  }
  vtk_polydata *pD = (vtk_polydata *)tr->data;

  //
  // we need to compute the distance to the camera for each cell.
  // Perform a dot product of each vertex with the supplied camera vector
  //

#ifdef USE_FLOAT_FOR_DEPTH_SORT
  // prepare an array for the distances
  pD->distances.resize(pD->points->size());
  // initialize our functor which will compute distance and store in a vector
  float3 cam = make_float3(cameravec[0], cameravec[1], cameravec[2]);

  distance_functor<float3> distance(cam);
  // apply distance functor using input and output arrays using zip_iterator
  thrust::for_each(
    thrust::make_zip_iterator(thrust::make_tuple(pD->points->begin(), pD->distances.begin())),
    thrust::make_zip_iterator(thrust::make_tuple(pD->points->end(),   pD->distances.end())),
    distance);
  //
  // To compute the average distance for each cell, we must
  // sum/gather 3 distances (one for each vertex) for every cell by
  // looking up the vertex indices from the cell array tuples
  //
  // prepare an array for the distances
  thrust::device_vector<float> cell_distances(pD->nCells);

  celldistance_functor<float> celldist(thrust::raw_pointer_cast(pD->distances.data()));

  thrust::for_each(
    thrust::make_zip_iterator(thrust::make_tuple(pD->cells->begin(), cell_distances.begin())),
    thrust::make_zip_iterator(thrust::make_tuple(pD->cells->end(),   cell_distances.end())),
    celldist);

  //
  // now we want to sort the cells using the average distance
  // we must copy the cell vertex index tuple during the sort
  //
  if (direction==0) {
    thrust::sort_by_key(cell_distances.begin(), cell_distances.end(), pD->cells->begin(), 
      thrust::greater<float>());
  }
  else {
    thrust::sort_by_key(cell_distances.begin(), cell_distances.end(), pD->cells->begin(), 
      thrust::less<float>());
  }

#else
  // prepare an array for the distances
  thrust::device_vector<double> distances(pD->points->size());
  // initialize our functor which will compute distance and store in a vector
  double3 cam = make_double3(cameravec[0], cameravec[1], cameravec[2]);

  distance_functor<double3> distance(cam);
  // apply distance functor using input and output arrays using zip_iterator
  thrust::for_each(
    thrust::make_zip_iterator(thrust::make_tuple(pD->points->begin(), distances.begin())),
    thrust::make_zip_iterator(thrust::make_tuple(pD->points->end(),   distances.end())),
    distance);

  //
  // To compute the average distance for each cell, we must
  // sum/gather 3 distances (one for each vertex) for every cell by
  // looking up the vertex indices from the cell array tuples
  //

  // prepare an array for the distances
  thrust::device_vector<double> cell_distances(pD->nCells);

  celldistance_functor<double> celldist(thrust::raw_pointer_cast(distances.data()));

  thrust::for_each(
    thrust::make_zip_iterator(thrust::make_tuple(pD->cells->begin(), cell_distances.begin())),
    thrust::make_zip_iterator(thrust::make_tuple(pD->cells->end(),   cell_distances.end())),
    celldist);

  //
  // now we want to sort the cells using the average distance
  // we must copy the cell vertex index tuple during the sort
  //
  if (direction==0) {
    thrust::sort_by_key(cell_distances.begin(), cell_distances.end(), pD->cells->begin(), 
      thrust::greater_equal<double>());
  }
  else {
    thrust::sort_by_key(cell_distances.begin(), cell_distances.end(), pD->cells->begin(), 
      thrust::less<double>());
  }
#endif

}

//------------------------------------------------------------------------------
void CudaTransferToGL(vtkPistonDataObject *id, unsigned long dataObjectMTimeCache,
                      hipGraphicsResource **vboResources,
                      unsigned char *colorptr,
                      double alpha,
                      bool &hasNormals, bool &hasColors, 
                      bool &useindexbuffers)
{
  vtkPistonReference *tr = id->GetReference();
  if (tr->type != VTK_POLY_DATA || tr->data == NULL) {
    // Type mismatch, don't bother trying
    return;
  }

  vtk_polydata *pD = (vtk_polydata *)tr->data;

  // Claim access to buffer for cuda
  hipError_t res;
  res = hipGraphicsMapResources(4, vboResources, 0);
  if (res != hipSuccess)
  {
    cerr << "Claim for CUDA failed ... " << hipGetErrorString(res) << endl;
    return;
  }

  size_t num_bytes;
  float3 *vertexBufferData;
  uint3  *cellsBufferData;
  float  *normalsBufferData;
  uchar4 *colorsBufferData; 

  res = hipGraphicsResourceGetMappedPointer((void **)&vertexBufferData, &num_bytes, vboResources[0]);
  if (res != hipSuccess) {
    cerr << "Get mappedpointer for vertices failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }
  res = hipGraphicsResourceGetMappedPointer((void **)&normalsBufferData, &num_bytes, vboResources[1]);
  if (res != hipSuccess) {
    cerr << "Get mappedpointer for normals failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }
  res = hipGraphicsResourceGetMappedPointer((void **)&colorsBufferData, &num_bytes, vboResources[2]);
  if (res != hipSuccess)
  {
    cerr << "Get mappedpointer for colors failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }

  res = hipGraphicsResourceGetMappedPointer((void **)&cellsBufferData, &num_bytes, vboResources[3]);
  if (res != hipSuccess)
  {
    std::string errormsg = hipGetErrorString(res);
    cerr << "Get mappedpointer for cell indices failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }

  // Copy on card verts to the shared on card gl buffer
  thrust::copy(pD->points->begin(), pD->points->end(),
               thrust::device_ptr<float3>(vertexBufferData));

  // Copy on card cell indices to the shared on card gl buffer
  if (pD->cells) {
    useindexbuffers = true;
    thrust::copy(pD->cells->begin(), pD->cells->end(),
                 thrust::device_ptr<uint3>(cellsBufferData));
  }

  hasNormals = false;
  if (pD->normals)
    {
    hasNormals = true;

    // Copy on card verts to the shared on card gl buffer
    thrust::copy(pD->normals->begin(), pD->normals->end(),
                 thrust::device_ptr<float>(normalsBufferData));
    }
  hasColors = false;


  if (pD->colors)
  {
    thrust::copy(pD->colors->begin(), pD->colors->end(), 
      thrust::device_ptr<uchar4>(colorsBufferData));
    hasColors = true;
  }
  else if (pD->distances.size()>0) {
    pD->colors->resize(pD->distances.size());
    float *raw_scalar = thrust::raw_pointer_cast(&pD->distances[0]);
    // Copy RGB values to GPU
    thrust::device_vector<unsigned char> onGPU(colorptr, colorptr+(256*4));
    unsigned char *raw_RGBA = thrust::raw_pointer_cast(&onGPU[0]);
    // create a lookuptable
    color_map colorMap(raw_RGBA, 256, 0, 2, 0.5, raw_scalar, NULL);
    // map all scalars through LUT
    thrust::transform(pD->distances.begin(), pD->distances.end(), pD->colors->begin(), colorMap);
  }
  else if (pD->scalars)
  {
/*
    std::vector<float> *colors = psc->ComputeScalarsColorsf();

    // Copy to GPU
    thrust::device_vector<float> onGPU(colors->begin(), colors->end());
    float *raw_ptr = thrust::raw_pointer_cast(&onGPU[0]);

    // Now run each scalar data through the map to choose a color for it

    double scalarRange[2];
    psc->GetScalarRange(scalarRange);

    float * opacitydata = pD->opacities ? thrust::raw_pointer_cast(pD->opacities->data()) : NULL;


  if (opacitydata==0) {
    color_map colorMap(raw_RGBA, rgba.size(), scalarRange[0], scalarRange[1], alpha, raw_scalar, raw_opacity);

    thrust::transform(scalardata.begin(), scalardata.end(), colours.begin(), colorMap);
  }
  else {
    // Now we'll create some zip_iterators for A and B
    auto first = thrust::make_zip_iterator(thrust::make_tuple(scalardata.begin(), opacitydata.begin()));
    auto last  = thrust::make_zip_iterator(thrust::make_tuple(scalardata.end(),   opacitydata.end()));

    color_map colorMap(raw_RGBA, rgba.size(), scalarRange[0], scalarRange[1], alpha, raw_scalar, raw_opacity);

    thrust::transform(first, last, colours.begin(), colorMap);
  }

    if (opacitydata) {
      // Now we'll create some zip_iterators for A and B
      Float2Iterator _first = thrust::make_zip_iterator(thrust::make_tuple(pD->scalars->begin(), pD->opacities->begin()));; 
      Float2Iterator  _last = thrust::make_zip_iterator(thrust::make_tuple(pD->scalars->end(),   pD->opacities->end()));

      color_map<FloatTuple> colorMap(raw_ptr, onGPU.size(), scalarRange[0], scalarRange[1], alpha, opacitydata);
      thrust::copy(thrust::make_transform_iterator(_first, colorMap),
                   thrust::make_transform_iterator(_last,  colorMap),
                   thrust::device_ptr<float4>(colorsBufferData));
    }
    else {
      color_map<float> colorMap(raw_ptr, onGPU.size(), scalarRange[0], scalarRange[1], alpha, opacitydata);
      thrust::copy(thrust::make_transform_iterator(pD->scalars->begin(), colorMap),
                   thrust::make_transform_iterator(pD->scalars->end(),   colorMap),
                   thrust::device_ptr<float4>(colorsBufferData));
    }
*/
  }

  // Allow GL to access again
  res = hipGraphicsUnmapResources(4, vboResources, 0);
  if (res != hipSuccess)
  {
    cerr << "Release from CUDA failed ... " << hipGetErrorString(res) << endl;
    return;
  }

  return;
}
//------------------------------------------------------------------------------
} //namespace
