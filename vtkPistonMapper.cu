#include "hip/hip_runtime.h"
#ifdef _WIN32
 #include <windows.h>
#endif

#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>

#include "vtkgl.h"
#include <cuda_gl_interop.h>

#include "piston/piston_math.h"
//
#include "vtkPistonDataObject.h"
#include "vtkPistonDataWrangling.h"
#include "vtkPistonReference.h"

#include "../vtkTwoScalarsToColorsPainter.h"

namespace vtkpiston {

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
typedef thrust::device_vector<float>::iterator FloatIterator;
typedef thrust::tuple<FloatIterator, FloatIterator> FloatIteratorTuple;
typedef thrust::tuple<float&, float&> FloatTuple;
typedef thrust::zip_iterator<FloatIteratorTuple> Float2Iterator;
//------------------------------------------------------------------------------
// The colour map struct is templated (T) over the iterator type that we will use
// in our transform function.
//
// Because we use complicated zip/tuple iterators which are partially dereferenced
// by the tuple/iterator code inside the transform call, we separately template
// the operator() over the final dereferenced (element) type that will be passed
// to the operator().
//
// Two specializations are provided for color tables with/without vertex opacity
// 1) scalars        : T=float(iterator), element=float 
// 2) scalar/opacity : T=FloatIteratorTuple(iterator), element=FloatTuple 

template<typename T>
struct color_map : public thrust::unary_function<T, float4>
{
  const float min;
  const float max;
  const int   size;
  float      *table;
  float       alpha;
  float      *opacity;

  color_map(float *table, int arrSize, float rMin, float rMax, double a, float *opacityarray) :
    min(rMin),
    max(rMax),
    size((arrSize / 3) - 1),
    table(table),
    alpha(a),
    opacity(opacityarray)
    {
    }
  
  // the internal calculation which is independent of template types
  __host__ __device__ inline float4 calc(float val, float opac) 
  { 
    // convert val to lookuptable index
    int index = 0;
    if ((max - min) > 0.0) {
      index = ( (val - min) / (max - min) ) * size;
    }
    if (index < 0)    index = 0;
    if (index > size) index = size;
    // convert to RGB tuple index
    index *= 3; 
    return make_float4(table[index], table[index + 1], table[index + 2], opac);
  };

  // declare an empty general templated functor operator which we will specialize later
  template<typename element>
  __host__ __device__ float4 operator()(element t) { 
    // should throw ("Error - this function must be specialized for the type used");
    return make_float4(1,1,1,1);
  }
};
//------------------------------------------------------------------------------
// doubly templated specialization
// overload the colormap operator() for tuple iterators which will hold <scalar, opacity>
template <> template<>
__host__ __device__ float4 color_map<FloatIteratorTuple>::operator()<FloatTuple>(FloatTuple t)
{
  float val  = thrust::get<0>(t);
  float opac = thrust::get<1>(t)*alpha;
  return calc(val, opac);
}
//------------------------------------------------------------------------------
// doubly templated specialization
// overload the colormap operator() for single color array
template <> template<>
__host__ __device__ float4 color_map<float>::operator()<float>(float t)
{
  float val = t;
  return calc(val, alpha);
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
void CudaGLInit()
{
  hipDeviceProp_t prop;
  int dev;

  // Fill it with zeros
  memset(&prop,0,sizeof(hipDeviceProp_t));

  // Pick a GPU capable of 1.0 or better
  prop.major=1; prop.minor=0;
  hipChooseDevice(&dev,&prop);

  // Set OpenGL device
  hipError_t res = cudaGLSetGLDevice(dev);

  if (res != hipSuccess)
    {
    std::cerr << "Set device failed  ... " << hipGetErrorString(res) << endl;
    return;
    }
}

//------------------------------------------------------------------------------
void CudaRegisterBuffer(struct hipGraphicsResource **vboResource,
                        GLuint vboBuffer)
{
  hipError_t res =
    hipGraphicsGLRegisterBuffer(vboResource, vboBuffer,
                                cudaGraphicsMapFlagsWriteDiscard);
  if (res != hipSuccess)
  {
    std::cerr << "Register buffer failed ... " << hipGetErrorString(res) << endl;
    return;
  }
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
struct distance_functor 
{
  float3 cameravector;

  // construct with a constant camera vector
  __host__ __device__ distance_functor(float3 &cam) : cameravector(cam) {}

  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<1>(t) = dot(thrust::get<0>(t), cameravector);  
  }
};
//------------------------------------------------------------------------------
struct celldistance_functor 
{
  const float *vertex_distances;
  
  // construct with a precomputed distance vector for every vertex
  __host__ __device__ celldistance_functor(float *v) : vertex_distances(v) {}
  
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  { 
    thrust::get<1>(t) = (vertex_distances[thrust::get<0>(t).x] + 
                         vertex_distances[thrust::get<0>(t).y] +
                         vertex_distances[thrust::get<0>(t).z])/3.0;
  }
};

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
void DepthSortPolygons(vtkPistonDataObject *id, double *cameravec, int direction)
{
  vtkPistonReference *tr = id->GetReference();
  if (tr->type != VTK_POLY_DATA || tr->data == NULL) {
    // Type mismatch, don't bother trying
    return;
  }
  vtk_polydata *pD = (vtk_polydata *)tr->data;

  //
  // we need to compute the distance to the camera for each cell.
  // Perform a dot product of each vertex with the supplied camera vector
  //

  // prepare an array for the distances
  thrust::device_vector<float> distances(pD->points->size());

  // initialize our functor which will compute distance and store in a vector
  float3 cam = make_float3(cameravec[0], cameravec[1], cameravec[2]);
  distance_functor distance(cam);

  // apply distance functor using input and output arrays using zip_iterator
  thrust::for_each(
    thrust::make_zip_iterator(thrust::make_tuple(pD->points->begin(), distances.begin())),
    thrust::make_zip_iterator(thrust::make_tuple(pD->points->end(),   distances.end())),
    distance);

  //
  // To compute the average distance for each cell, we must
  // sum/gather 3 distances (one for each vertex) for every cell by
  // looking up the vertex indices from the cell array tuples
  //

  // prepare an array for the distances
  thrust::device_vector<float> cell_distances(pD->nCells);

  celldistance_functor celldist(thrust::raw_pointer_cast(distances.data()));

  thrust::for_each(
    thrust::make_zip_iterator(thrust::make_tuple(pD->cells->begin(), cell_distances.begin())),
    thrust::make_zip_iterator(thrust::make_tuple(pD->cells->end(),   cell_distances.end())),
    celldist);

  //
  // now we want to sort the cells using the average distance
  // we must copy the cell vertex index tuple during the sort
  //
  if (direction==0) {
    thrust::sort_by_key(cell_distances.begin(), cell_distances.end(), pD->cells->begin(), 
      thrust::greater<float>());
  }
  else {
    thrust::sort_by_key(cell_distances.begin(), cell_distances.end(), pD->cells->begin(), 
      thrust::less<float>());
  }
}

//------------------------------------------------------------------------------
void CudaTransferToGL(vtkPistonDataObject *id, unsigned long dataObjectMTimeCache,
                      vtkTwoScalarsToColorsPainter *psc,
                      hipGraphicsResource **vboResources,
                      double alpha,
                      bool &hasNormals, bool &hasColors, 
                      bool &useindexbuffers)
{
  vtkPistonReference *tr = id->GetReference();
  if (tr->type != VTK_POLY_DATA || tr->data == NULL)
    {
    // Type mismatch, don't bother trying
    return;
    }

  vtk_polydata *pD = (vtk_polydata *)tr->data;

  // Claim access to buffer for cuda
  hipError_t res;
  res = hipGraphicsMapResources(4, vboResources, 0);
  if (res != hipSuccess)
  {
    cerr << "Claim for CUDA failed ... " << hipGetErrorString(res) << endl;
    return;
  }

  size_t num_bytes;
  float3 *vertexBufferData;
  uint3  *cellsBufferData;
  float  *normalsBufferData;
  float4 *colorsBufferData; 

  res = hipGraphicsResourceGetMappedPointer
      ((void **)&vertexBufferData, &num_bytes, vboResources[0]);
  if(res != hipSuccess) {
    cerr << "Get mappedpointer for vertices failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }
  res = hipGraphicsResourceGetMappedPointer
      ((void **)&normalsBufferData, &num_bytes, vboResources[1]);
  if(res != hipSuccess) {
    cerr << "Get mappedpointer for normals failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }
  res = hipGraphicsResourceGetMappedPointer
      ((void **)&colorsBufferData, &num_bytes, vboResources[2]);
  if(res != hipSuccess)
  {
    cerr << "Get mappedpointer for colors failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }

  res = hipGraphicsResourceGetMappedPointer
      ((void **)&cellsBufferData, &num_bytes, vboResources[3]);
  if(res != hipSuccess)
  {
    std::string errormsg = hipGetErrorString(res);
    cerr << "Get mappedpointer for cell indices failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }

  // Copy on card verts to the shared on card gl buffer
  thrust::copy(pD->points->begin(), pD->points->end(),
               thrust::device_ptr<float3>(vertexBufferData));

  // Copy on card cell indices to the shared on card gl buffer
  if (pD->cells) {
    useindexbuffers = true;
    thrust::copy(pD->cells->begin(), pD->cells->end(),
                 thrust::device_ptr<uint3>(cellsBufferData));
  }

  hasNormals = false;
  if (pD->normals)
    {
    hasNormals = true;

    // Copy on card verts to the shared on card gl buffer
    thrust::copy(pD->normals->begin(), pD->normals->end(),
                 thrust::device_ptr<float>(normalsBufferData));
    }
  hasColors = false;


  if (0 && pD->colors)
  {
    thrust::copy(pD->colors->begin(), pD->colors->end(), 
      thrust::device_ptr<float4>(colorsBufferData));
  }
  else if (pD->scalars)
  {
/*
    double scalarRange[2];
    id->GetScalarsRange(scalarRange);

    hasColors = true;

    if(id->GetMTime() > dataObjectMTimeCache)
      {
      vtkPiston::minmax_pair<float> result = vtkPiston::find_min_max(
                                              pD->scalars);

      scalarRange[0] = static_cast<double>(result.min_val);
      scalarRange[1] = static_cast<double>(result.max_val);

      // Set parameters to compute scalars colors
      const int numvalues = 256;
      id->SetScalarsRange(scalarRange);
      psc->SetTableRange(scalarRange[0], scalarRange[1]);
      psc->SetNumberOfValues(numvalues);
      }
*/

    std::vector<float> *colors = psc->ComputeScalarsColorsf();

    // Copy to GPU
    thrust::device_vector<float> onGPU(colors->begin(), colors->end());
    float *raw_ptr = thrust::raw_pointer_cast(&onGPU[0]);

    // Now run each scalar data through the map to choose a color for it

    double scalarRange[2];
    psc->GetScalarRange(scalarRange);

    float * opacitydata = pD->opacities ? thrust::raw_pointer_cast(pD->opacities->data()) : NULL;

    if (opacitydata) {
      // Now we'll create some zip_iterators for A and B
      Float2Iterator _first = thrust::make_zip_iterator(thrust::make_tuple(pD->scalars->begin(), pD->opacities->begin()));; 
      Float2Iterator  _last = thrust::make_zip_iterator(thrust::make_tuple(pD->scalars->end(),   pD->opacities->end()));

      color_map<FloatIteratorTuple> colorMap(raw_ptr, onGPU.size(), scalarRange[0], scalarRange[1], alpha, opacitydata);
      thrust::copy(thrust::make_transform_iterator(_first, colorMap),
                   thrust::make_transform_iterator(_last,  colorMap),
                   thrust::device_ptr<float4>(colorsBufferData));
    }
    else {
      color_map<float> colorMap(raw_ptr, onGPU.size(), scalarRange[0], scalarRange[1], alpha, opacitydata);
      thrust::copy(thrust::make_transform_iterator(pD->scalars->begin(), colorMap),
                   thrust::make_transform_iterator(pD->scalars->end(),   colorMap),
                   thrust::device_ptr<float4>(colorsBufferData));
    }
  }

  // Allow GL to access again
  res = hipGraphicsUnmapResources(4, vboResources, 0);
  if (res != hipSuccess)
  {
    cerr << "Release from CUDA failed ... " << hipGetErrorString(res) << endl;
    return;
  }

  return;
}
//------------------------------------------------------------------------------
} //namespace
